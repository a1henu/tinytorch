#include "hip/hip_runtime.h"
/**
 * @file relu.cu
 * @brief relu operator implementation for GPU
 * 
 * @copyright Copyright (c) 2024 chenxu bai
 * Licensed under the MIT License.
 */

#include "macros.h"

template <typename Tp>
__global__ void 
kernel_relu_f(Tp* output, Tp* input, size_t size) {
    CUDA_KERNEL_LOOP(i, size) {
        output[i] = input[i] > 0 ? input[i] : 0;
    }
}

template <typename Tp>
__global__ void 
kernel_relu_b(Tp* output, Tp* input, Tp* grad, size_t size) {
    CUDA_KERNEL_LOOP(i, size) {
        output[i] = input[i] > 0 ? grad[i] : 0;
    }
}


template <typename Tp>
void relu_forward(Tp* output, Tp* input, size_t size) {
    kernel_relu_f<Tp><<<CUDA_GET_BLOCKS(size), CUDA_K_THREADS>>>(output, input, size);
}

template <typename Tp>
void relu_backward(Tp* output, Tp* input, Tp* grad, size_t size) {
    kernel_relu_b<Tp><<<CUDA_GET_BLOCKS(size), CUDA_K_THREADS>>>(output, input, grad, size);
}