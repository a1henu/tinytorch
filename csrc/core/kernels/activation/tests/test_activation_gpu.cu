/**
 * @file test_activation_gpu.cu
 * @brief Activation function test cases for GPU
 * 
 * @copyright Copyright (c) 2024 chenxu bai
 * Licensed under the MIT License.
 */

#include <gtest/gtest.h>
#include <vector>

#include "core/device/device.h"
#include "core/kernels/activation/relu.h"
#include "core/kernels/activation/sigmoid.h"

class TestReLU : public ::testing::Test {
protected:
    std::vector<double> v;
    std::vector<double> g;

    std::vector<double> v_relu_f;
    std::vector<double> v_relu_b;

    double* v_g;
    double* g_g;
    double* v_relu_f_g;
    double* v_relu_b_g;

    int v_dim;

    void SetUp() override {
        v = {
            0.021779, -0.091378, 2.529141, -1.314787, 2.063127, 
            0.499841, 0.930395, 1.301085, 3.620728, -0.509598, 
            -0.729976, 1.701381, -0.519704, 0.361764, -0.010430, 
            0.764627, 0.749973, 0.889580, 0.072533, 0.252502, 
            0.179101, 2.111640, 0.788848, -0.130065, 1.355981, 
            0.541689, 0.206137, 1.232952, 0.943046, -0.229882
        };
        g = {
            0.611874, 0.043436, 0.710300, -0.144175, 0.307186, 
            -0.469658, 0.082649, -0.185967, -1.892874, -1.613372, 
            1.128987, -1.476294, 0.511537, 2.049930, 0.040707, 
            -0.332097, 0.460975, 0.286529, -0.167816, -1.922494, 
            0.093031, -0.290727, 0.196690, -0.644933, -0.173954, 
            -0.749864, 0.717477, -0.122634, 0.127579, 2.801707
        };
        v_relu_f = {
            0.021779, 0.000000, 2.529141, 0.000000, 2.063127, 
            0.499841, 0.930395, 1.301085, 3.620728, 0.000000, 
            0.000000, 1.701381, 0.000000, 0.361764, 0.000000, 
            0.764627, 0.749973, 0.889580, 0.072533, 0.252502, 
            0.179101, 2.111640, 0.788848, 0.000000, 1.355981, 
            0.541689, 0.206137, 1.232952, 0.943046, 0.000000
        };
        v_relu_b = {
            0.611874, 0.000000, 0.710300, 0.000000, 0.307186, 
            -0.469658, 0.082649, -0.185967, -1.892874, 0.000000, 
            0.000000, -1.476294, 0.000000, 2.049930, 0.000000, 
            -0.332097, 0.460975, 0.286529, -0.167816, -1.922494, 
            0.093031, -0.290727, 0.196690, 0.000000, -0.173954, 
            -0.749864, 0.717477, -0.122634, 0.127579, 0.000000
        };
        v_dim = v.size();

        hipMalloc(&v_g, v_dim * sizeof(double));
        hipMalloc(&g_g, v_dim * sizeof(double));
        hipMalloc(&v_relu_f_g, v_dim * sizeof(double));
        hipMalloc(&v_relu_b_g, v_dim * sizeof(double));

        hipMemcpy(v_g, v.data(), v_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(g_g, g.data(), v_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(v_relu_f_g, v_relu_f.data(), v_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(v_relu_b_g, v_relu_b.data(), v_dim * sizeof(double), hipMemcpyHostToDevice);
    }
    void TearDown() override {
        hipFree(v_g);
        hipFree(g_g);
        hipFree(v_relu_f_g);
        hipFree(v_relu_b_g);
    }

};

class TestSigmoid : public ::testing::Test {
protected:
    std::vector<double> x;
    std::vector<double> g;

    std::vector<double> x_sigmoid_f;
    std::vector<double> x_sigmoid_b;

    double* x_g;
    double* g_g;
    double* x_sigmoid_f_g;
    double* x_sigmoid_b_g;

    int x_dim;

    void SetUp() override {
        x = {
            0.760609, -0.715157, 0.048647, -0.090885, 0.849236, 
            1.422869, 1.486288, 1.030767, 0.924290, -1.496499, 
            1.142868, 0.366450, 0.224117, 2.006245, 0.116255, 
            0.295170, -1.305522, 0.590613, -0.296552, -0.988433, 
            -1.320542, 0.617037, 0.462768, -1.011641, -0.022526, 
            -0.207176, 1.544988, -0.053472, -0.179526, -1.048691
        };
        g = {
            0.611874, 0.043436, 0.710300, -0.144175, 0.307186, 
            -0.469658, 0.082649, -0.185967, -1.892874, -1.613372, 
            1.128987, -1.476294, 0.511537, 2.049930, 0.040707, 
            -0.332097, 0.460975, 0.286529, -0.167816, -1.922494, 
            0.093031, -0.290727, 0.196690, -0.644933, -0.173954, 
            -0.749864, 0.717477, -0.122634, 0.127579, 2.801707
        };
        x_sigmoid_f = {
            0.681486, 0.328460, 0.512159, 0.477294, 0.700407, 
            0.805788, 0.815521, 0.737065, 0.715915, 0.182948, 
            0.758206, 0.590601, 0.555796, 0.881451, 0.529031, 
            0.573261, 0.213237, 0.643506, 0.426401, 0.271222, 
            0.210728, 0.649544, 0.613671, 0.266659, 0.494369, 
            0.448391, 0.824189, 0.486635, 0.455239, 0.259476
        };
        x_sigmoid_b = {
            0.132815, 0.009581, 0.177470, -0.035969, 0.064459, 
            -0.073499, 0.012434, -0.036041, -0.384974, -0.241164, 
            0.206977, -0.356955, 0.126292, 0.214207, 0.010142, 
            -0.081242, 0.077336, 0.065731, -0.041045, -0.380001, 
            0.015473, -0.066180, 0.046631, -0.126118, -0.043483, 
            -0.185469, 0.103964, -0.030637, 0.031639, 0.538344
        };
        x_dim = x.size();

        hipMalloc(&x_g, x_dim * sizeof(double));
        hipMalloc(&g_g, x_dim * sizeof(double));
        hipMalloc(&x_sigmoid_f_g, x_dim * sizeof(double));
        hipMalloc(&x_sigmoid_b_g, x_dim * sizeof(double));

        hipMemcpy(x_g, x.data(), x_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(g_g, g.data(), x_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(x_sigmoid_f_g, x_sigmoid_f.data(), x_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(x_sigmoid_b_g, x_sigmoid_b.data(), x_dim * sizeof(double), hipMemcpyHostToDevice);
    }
    void TearDown() override {
        hipFree(x_g);
        hipFree(g_g);
        hipFree(x_sigmoid_f_g);
        hipFree(x_sigmoid_b_g);
    }

};

TEST_F(TestReLU, forward) {
    double* vt_relu_f_g;
    hipMalloc(&vt_relu_f_g, v_dim * sizeof(double));
    activation::relu_forward<double, device::GPU>()(device::gpu_device, vt_relu_f_g, v_g, v_dim);

    double* vt_relu_f = new double[v_dim];
    hipMemcpy(vt_relu_f, vt_relu_f_g, v_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < v_dim; i++) {
        EXPECT_NEAR(vt_relu_f[i], v_relu_f[i], 1e-6);
    }
}

TEST_F(TestReLU, backward) {
    double* vt_relu_b_g;
    hipMalloc(&vt_relu_b_g, v_dim * sizeof(double));
    activation::relu_backward<double, device::GPU>()(device::gpu_device, vt_relu_b_g, v_g, g_g, v_dim);

    double* vt_relu_b = new double[v_dim];
    hipMemcpy(vt_relu_b, vt_relu_b_g, v_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < v_dim; i++) {
        EXPECT_NEAR(vt_relu_b[i], v_relu_b[i], 1e-6);
    }
}

TEST_F(TestSigmoid, forward) {
    double* xt_sigmoid_f_g;
    hipMalloc(&xt_sigmoid_f_g, x_dim * sizeof(double));
    activation::sigmoid_forward<double, device::GPU>()(device::gpu_device, xt_sigmoid_f_g, x_g, x_dim);

    double* xt_sigmoid_f = new double[x_dim];
    hipMemcpy(xt_sigmoid_f, xt_sigmoid_f_g, x_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < x_dim; i++) {
        EXPECT_NEAR(xt_sigmoid_f[i], x_sigmoid_f[i], 1e-6);
    }
}

TEST_F(TestSigmoid, backward) {
    double* xt_sigmoid_b_g;
    hipMalloc(&xt_sigmoid_b_g, x_dim * sizeof(double));
    activation::sigmoid_backward<double, device::GPU>()(device::gpu_device, xt_sigmoid_b_g, x_g, g_g, x_dim);

    double* xt_sigmoid_b = new double[x_dim];
    hipMemcpy(xt_sigmoid_b, xt_sigmoid_b_g, x_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < x_dim; i++) {
        EXPECT_NEAR(xt_sigmoid_b[i], x_sigmoid_b[i], 1e-6);
    }
}

int main(int argc, char** argv) {
    std::cout << "run test for CORE::KERNELS::ACTIVATION::CPU" << std::endl << std::endl;
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}

