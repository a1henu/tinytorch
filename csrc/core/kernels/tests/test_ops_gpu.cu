/**
 * @file test_ops_gpu.cu
 * @brief Math operator test cases for GPU
 * 
 * @copyright Copyright (c) 2024 chenxu bai
 * Licensed under the MIT License.
 */

#include <gtest/gtest.h>
#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
#include <iterator>

#include "core/device/device.h"
#include "core/kernels/ops.h"

#include "error/error.h"

std::vector<float> generate_random_vector(size_t size, float min_value, float max_value) {
    std::vector<float> vec(size);
    std::random_device rd;  
    std::mt19937 gen(rd()); 
    std::uniform_real_distribution<> dis(min_value, max_value); 

    std::generate(vec.begin(), vec.end(), [&]() { return dis(gen); });

    return vec;
}

std::vector<double> generate_random_vector(size_t size, double min_value, double max_value) {
    std::vector<double> vec(size);
    std::random_device rd;  
    std::mt19937 gen(rd()); 
    std::uniform_real_distribution<> dis(min_value, max_value); 

    std::generate(vec.begin(), vec.end(), [&]() { return dis(gen); });

    return vec;
}

class TestOps : public ::testing::Test {
protected:
    std::vector<double> v1;
    std::vector<double> v2;

    double* vt_1;
    double* vt_2;

    int vt_dim;

    void SetUp() override {
        v1 = generate_random_vector(100, 0.0, 1.0); 
        v2 = generate_random_vector(100, 0.0, 1.0);
        vt_dim = v1.size();

        hipMalloc(&vt_1, vt_dim * sizeof(double));
        hipMalloc(&vt_2, vt_dim * sizeof(double));

        hipMemcpy(vt_1, v1.data(), vt_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(vt_2, v2.data(), vt_dim * sizeof(double), hipMemcpyHostToDevice);
    }
    void TearDown() override {
        hipFree(vt_1);
        hipFree(vt_2);
    }
    
    using add_cpu_op = ops::add_op<double, device::CPU>;
    using sub_cpu_op = ops::sub_op<double, device::CPU>;
    using smatmul_cpu_op = ops::matmul_op<float, device::CPU>;
    using dmatmul_cpu_op = ops::matmul_op<double, device::CPU>;
    using equal_cpu_op = ops::equal_op<double, device::CPU>;
    using ones_cpu_op = ops::ones_op<double, device::CPU>;
    using eye_cpu_op = ops::eye_op<double, device::CPU>;

    using add_gpu_op = ops::add_op<double, device::GPU>;
    using sub_gpu_op = ops::sub_op<double, device::GPU>;
    using smatmul_gpu_op = ops::matmul_op<float, device::GPU>;
    using dmatmul_gpu_op = ops::matmul_op<double, device::GPU>;
    using equal_gpu_op = ops::equal_op<double, device::GPU>;
    using ones_gpu_op = ops::ones_op<double, device::GPU>;
    using eye_gpu_op = ops::eye_op<double, device::GPU>;
};

TEST_F(TestOps, TestAddOp_gpu_1) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    add_gpu_op()(device::gpu_device, vt_out, vt_1, vt_2, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] + v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestOps, TestAddOp_gpu_2) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    add_gpu_op()(device::gpu_device, vt_out, vt_2, vt_1, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] + v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestOps, TestSubOp_gpu) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    sub_gpu_op()(device::gpu_device, vt_out, vt_1, vt_2, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] - v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestOps, TestMatmulOp_gpu_float) {
    const int m = 30, n = 40, k = 35;
    std::vector<float> A = generate_random_vector(m * k, 0.0f, 1.0f);
    std::vector<float> B = generate_random_vector(k * n, 0.0f, 1.0f);
    float* d_A;
    float* d_B;
    float* d_C;
    float* h_C = new float[m * n];

    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, A.data(), m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), k * n * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    smatmul_gpu_op()(device::gpu_device, "N", "N", m, n, k, alpha, d_A, k, d_B, n, beta, d_C, n);

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<float> C_expected(m * n, 0.0f);
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            for (int p = 0; p < k; ++p) {
                C_expected[i * n + j] += A[i * k + p] * B[p * n + j];
            }
        }
    }

    for (int i = 0; i < m * n; ++i) {
        EXPECT_NEAR(h_C[i], C_expected[i], 1e-4f);
    }

    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

TEST_F(TestOps, TestMatmulOp_gpu_double) {
    const int m = 2, n = 3, k = 4;
    std::vector<double> A = generate_random_vector(m * k, 0.0, 1.0);
    std::vector<double> B = generate_random_vector(k * n, 0.0, 1.0);
    double* d_A;
    double* d_B;
    double* d_C;
    double* h_C = new double[m * n];

    hipMalloc(&d_A, m * k * sizeof(double));
    hipMalloc(&d_B, k * n * sizeof(double));
    hipMalloc(&d_C, m * n * sizeof(double));

    hipMemcpy(d_A, A.data(), m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), k * n * sizeof(double), hipMemcpyHostToDevice);

    const double alpha = 1.0;
    const double beta = 0.0;

    dmatmul_gpu_op()(device::gpu_device, "N", "N", m, n, k, alpha, d_A, k, d_B, n, beta, d_C, n);

    hipMemcpy(h_C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    std::vector<double> C_expected(m * n, 0.0);
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            for (int p = 0; p < k; ++p) {
                C_expected[i * n + j] += A[i * k + p] * B[p * n + j];
            }
        }
    }

    for (int i = 0; i < m * n; ++i) {
        EXPECT_NEAR(h_C[i], C_expected[i], 1e-4);
    }

    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

TEST_F(TestOps, TestEqualOp_gpu_1) {
    bool vt_out_c = true;
    bool* vt_out_g;
    hipMalloc(&vt_out_g, sizeof(bool));
    equal_gpu_op()(device::gpu_device, vt_out_g, vt_1, vt_2, vt_dim);
    hipMemcpy(&vt_out_c, vt_out_g, sizeof(bool), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        if (v1[i] != v2[i]) {
            EXPECT_FALSE(vt_out_c);
            hipFree(vt_out_g);
            return;
        }
    }
    EXPECT_TRUE(vt_out_c);
    hipFree(vt_out_g);
}

TEST_F(TestOps, TestEqualOp_gpu_2) {
    double* vt_out1;
    double* vt_out2;
    bool* vt_out_g;
    bool* vt_out_c = new bool;
    hipMalloc(&vt_out1, vt_dim * sizeof(double));
    hipMalloc(&vt_out2, vt_dim * sizeof(double));
    hipMalloc(&vt_out_g, sizeof(bool));
    add_gpu_op()(device::gpu_device, vt_out1, vt_1, vt_2, vt_dim);
    add_gpu_op()(device::gpu_device, vt_out2, vt_2, vt_1, vt_dim);
    equal_gpu_op()(device::gpu_device, vt_out_g, vt_out1, vt_out2, vt_dim);
    hipMemcpy(vt_out_c, vt_out_g, sizeof(bool), hipMemcpyDeviceToHost);
    EXPECT_TRUE(*vt_out_c);
    hipFree(vt_out1);
    hipFree(vt_out2);
    hipFree(vt_out_g);
    delete vt_out_c;
}

TEST_F(TestOps, TestOnesOp_gpu) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    ones_gpu_op()(device::gpu_device, vt_out, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], 1.0);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestOps, TestEyeOp_gpu) {
    const int dim = 100;
    double* vt_out;
    double* vt_out_cpu = new double[dim * dim];
    hipMalloc(&vt_out, dim * dim * sizeof(double));
    eye_gpu_op()(device::gpu_device, vt_out, dim);
    hipMemcpy(vt_out_cpu, vt_out, dim * dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < dim; ++i) {
        for (int j = 0; j < dim; ++j) {
            if (i == j) {
                EXPECT_EQ(vt_out_cpu[i * dim + j], 1.0);
            } else {
                EXPECT_EQ(vt_out_cpu[i * dim + j], 0.0);
            }
        }
    }
}

int main(int argc, char** argv) {
std::cout << "run test for CORE::KERNELS::OPS::GPU" << std::endl << std::endl;
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
