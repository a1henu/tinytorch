/**
 * @file test_ops_gpu.cu
 * @brief Math operator test cases for GPU
 * 
 * @copyright Copyright (c) 2024 chenxu bai
 * Licensed under the MIT License.
 */

#include <gtest/gtest.h>
#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
#include <iterator>

#include "core/device/device.h"
#include "core/kernels/ops.h"

#include "error/error.h"

std::vector<double> generate_random_vector(size_t size, double min_value, double max_value) {
    std::vector<double> vec(size);
    std::random_device rd;  
    std::mt19937 gen(rd()); 
    std::uniform_real_distribution<> dis(min_value, max_value); 

    std::generate(vec.begin(), vec.end(), [&]() { return dis(gen); });

    return vec;
}

class TestMemory : public ::testing::Test {
protected:
    std::vector<double> v1;
    std::vector<double> v2;

    double* vt_1;
    double* vt_2;

    int vt_dim;

    void SetUp() override {
        v1 = generate_random_vector(100, 0.0, 1.0); 
        v2 = generate_random_vector(100, 0.0, 1.0);
        vt_dim = v1.size();

        hipMalloc(&vt_1, vt_dim * sizeof(double));
        hipMalloc(&vt_2, vt_dim * sizeof(double));

        hipMemcpy(vt_1, v1.data(), vt_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(vt_2, v2.data(), vt_dim * sizeof(double), hipMemcpyHostToDevice);
    }
    void TearDown() override {
        hipFree(vt_1);
        hipFree(vt_2);
    }
    
    using add_cpu_op = ops::add_op<double, device::CPU>;
    using sub_cpu_op = ops::sub_op<double, device::CPU>;
    using equal_cpu_op = ops::equal_op<double, device::CPU>;

    using add_gpu_op = ops::add_op<double, device::GPU>;
    using sub_gpu_op = ops::sub_op<double, device::GPU>;
    using equal_gpu_op = ops::equal_op<double, device::GPU>;
};

TEST_F(TestMemory, TestAddOp_gpu_1) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    add_gpu_op()(device::gpu_device, vt_out, vt_1, vt_2, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] + v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestMemory, TestAddOp_gpu_2) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    add_gpu_op()(device::gpu_device, vt_out, vt_2, vt_1, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] + v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestMemory, TestSubOp_gpu) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    sub_gpu_op()(device::gpu_device, vt_out, vt_1, vt_2, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] - v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestMemory, TestEqualOp_gpu_1) {
    bool vt_out_c = true;
    bool* vt_out_g;
    hipMalloc(&vt_out_g, sizeof(bool));
    equal_gpu_op()(device::gpu_device, vt_out_g, vt_1, vt_2, vt_dim);
    hipMemcpy(&vt_out_c, vt_out_g, sizeof(bool), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        if (v1[i] != v2[i]) {
            EXPECT_FALSE(vt_out_c);
            hipFree(vt_out_g);
            return;
        }
    }
    EXPECT_TRUE(vt_out_c);
    hipFree(vt_out_g);
}

TEST_F(TestMemory, TestEqualOp_gpu_2) {
    double* vt_out1;
    double* vt_out2;
    bool* vt_out_g;
    bool* vt_out_c = new bool;
    hipMalloc(&vt_out1, vt_dim * sizeof(double));
    hipMalloc(&vt_out2, vt_dim * sizeof(double));
    hipMalloc(&vt_out_g, sizeof(bool));
    add_gpu_op()(device::gpu_device, vt_out1, vt_1, vt_2, vt_dim);
    add_gpu_op()(device::gpu_device, vt_out2, vt_2, vt_1, vt_dim);
    equal_gpu_op()(device::gpu_device, vt_out_g, vt_out1, vt_out2, vt_dim);
    hipMemcpy(vt_out_c, vt_out_g, sizeof(bool), hipMemcpyDeviceToHost);
    EXPECT_TRUE(*vt_out_c);
    hipFree(vt_out1);
    hipFree(vt_out2);
    hipFree(vt_out_g);
    delete vt_out_c;
}

int main(int argc, char** argv) {
std::cout << "run test for CORE::KERNELS::OPS::GPU" << std::endl << std::endl;
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
