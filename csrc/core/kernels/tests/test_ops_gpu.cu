/**
 * @file test_ops_gpu.cu
 * @brief Math operator test cases for GPU
 * 
 * @copyright Copyright (c) 2024 chenxu bai
 * Licensed under the MIT License.
 */

#include <gtest/gtest.h>
#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
#include <iterator>

#include "core/device/device.h"
#include "core/kernels/ops.h"

#include "error/error.h"

std::vector<float> generate_random_vector(size_t size, float min_value, float max_value) {
    std::vector<float> vec(size);
    std::random_device rd;  
    std::mt19937 gen(rd()); 
    std::uniform_real_distribution<> dis(min_value, max_value); 

    std::generate(vec.begin(), vec.end(), [&]() { return dis(gen); });

    return vec;
}

std::vector<double> generate_random_vector(size_t size, double min_value, double max_value) {
    std::vector<double> vec(size);
    std::random_device rd;  
    std::mt19937 gen(rd()); 
    std::uniform_real_distribution<> dis(min_value, max_value); 

    std::generate(vec.begin(), vec.end(), [&]() { return dis(gen); });

    return vec;
}

class TestOps : public ::testing::Test {
protected:
    std::vector<double> v1;
    std::vector<double> v2;

    double* vt_1;
    double* vt_2;

    int vt_dim;

    void SetUp() override {
        v1 = generate_random_vector(100, 0.0, 1.0); 
        v2 = generate_random_vector(100, 0.0, 1.0);
        vt_dim = v1.size();

        hipMalloc(&vt_1, vt_dim * sizeof(double));
        hipMalloc(&vt_2, vt_dim * sizeof(double));

        hipMemcpy(vt_1, v1.data(), vt_dim * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(vt_2, v2.data(), vt_dim * sizeof(double), hipMemcpyHostToDevice);
    }
    void TearDown() override {
        hipFree(vt_1);
        hipFree(vt_2);
    }
    
    using add_cpu_op = ops::add_op<double, device::CPU>;
    using sub_cpu_op = ops::sub_op<double, device::CPU>;
    using smatmul_cpu_op = ops::matmul_op<float, device::CPU>;
    using dmatmul_cpu_op = ops::matmul_op<double, device::CPU>;
    using equal_cpu_op = ops::equal_op<double, device::CPU>;
    using ones_cpu_op = ops::ones_op<double, device::CPU>;
    using eye_cpu_op = ops::eye_op<double, device::CPU>;
    using im2col_cpu_op = ops::im2col_op<int, device::CPU>;
    using max_pool_cpu_op = ops::max_pool_forward_op<double, device::CPU>;
    using max_pool_backward_cpu_op = ops::max_pool_backward_op<double, device::CPU>;

    using add_gpu_op = ops::add_op<double, device::GPU>;
    using sub_gpu_op = ops::sub_op<double, device::GPU>;
    using smatmul_gpu_op = ops::matmul_op<float, device::GPU>;
    using dmatmul_gpu_op = ops::matmul_op<double, device::GPU>;
    using equal_gpu_op = ops::equal_op<double, device::GPU>;
    using ones_gpu_op = ops::ones_op<double, device::GPU>;
    using eye_gpu_op = ops::eye_op<double, device::GPU>;
    using im2col_gpu_op = ops::im2col_op<int, device::GPU>;
    using max_pool_gpu_op = ops::max_pool_forward_op<double, device::GPU>;
    using max_pool_backward_gpu_op = ops::max_pool_backward_op<double, device::GPU>;
};

TEST_F(TestOps, TestAddOp_gpu_1) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    add_gpu_op()(device::gpu_device, vt_out, vt_1, vt_2, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] + v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestOps, TestAddOp_gpu_2) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    add_gpu_op()(device::gpu_device, vt_out, vt_2, vt_1, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] + v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestOps, TestSubOp_gpu) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    sub_gpu_op()(device::gpu_device, vt_out, vt_1, vt_2, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], v1[i] - v2[i]);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestOps, TestMatmulOp_gpu_float) {
    const int m = 30, n = 40, k = 35;
    std::vector<float> A = generate_random_vector(m * k, 0.0f, 1.0f);
    std::vector<float> B = generate_random_vector(k * n, 0.0f, 1.0f);
    float* d_A;
    float* d_B;
    float* d_C;
    float* h_C = new float[m * n];

    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, A.data(), m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), k * n * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    smatmul_gpu_op()(device::gpu_device, "N", "N", m, n, k, alpha, d_A, k, d_B, n, beta, d_C, n);

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<float> C_expected(m * n, 0.0f);
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            for (int p = 0; p < k; ++p) {
                C_expected[i * n + j] += A[i * k + p] * B[p * n + j];
            }
        }
    }

    for (int i = 0; i < m * n; ++i) {
        EXPECT_NEAR(h_C[i], C_expected[i], 1e-4f);
    }

    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

TEST_F(TestOps, TestMatmulOp_gpu_double) {
    const int m = 2, n = 3, k = 4;
    std::vector<double> A = generate_random_vector(m * k, 0.0, 1.0);
    std::vector<double> B = generate_random_vector(k * n, 0.0, 1.0);
    double* d_A;
    double* d_B;
    double* d_C;
    double* h_C = new double[m * n];

    hipMalloc(&d_A, m * k * sizeof(double));
    hipMalloc(&d_B, k * n * sizeof(double));
    hipMalloc(&d_C, m * n * sizeof(double));

    hipMemcpy(d_A, A.data(), m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), k * n * sizeof(double), hipMemcpyHostToDevice);

    const double alpha = 1.0;
    const double beta = 0.0;

    dmatmul_gpu_op()(device::gpu_device, "N", "N", m, n, k, alpha, d_A, k, d_B, n, beta, d_C, n);

    hipMemcpy(h_C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    std::vector<double> C_expected(m * n, 0.0);
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            for (int p = 0; p < k; ++p) {
                C_expected[i * n + j] += A[i * k + p] * B[p * n + j];
            }
        }
    }

    for (int i = 0; i < m * n; ++i) {
        EXPECT_NEAR(h_C[i], C_expected[i], 1e-4);
    }

    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

TEST_F(TestOps, TestEqualOp_gpu_1) {
    bool vt_out_c = true;
    bool* vt_out_g;
    hipMalloc(&vt_out_g, sizeof(bool));
    equal_gpu_op()(device::gpu_device, vt_out_g, vt_1, vt_2, vt_dim);
    hipMemcpy(&vt_out_c, vt_out_g, sizeof(bool), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        if (v1[i] != v2[i]) {
            EXPECT_FALSE(vt_out_c);
            hipFree(vt_out_g);
            return;
        }
    }
    EXPECT_TRUE(vt_out_c);
    hipFree(vt_out_g);
}

TEST_F(TestOps, TestEqualOp_gpu_2) {
    double* vt_out1;
    double* vt_out2;
    bool* vt_out_g;
    bool* vt_out_c = new bool;
    hipMalloc(&vt_out1, vt_dim * sizeof(double));
    hipMalloc(&vt_out2, vt_dim * sizeof(double));
    hipMalloc(&vt_out_g, sizeof(bool));
    add_gpu_op()(device::gpu_device, vt_out1, vt_1, vt_2, vt_dim);
    add_gpu_op()(device::gpu_device, vt_out2, vt_2, vt_1, vt_dim);
    equal_gpu_op()(device::gpu_device, vt_out_g, vt_out1, vt_out2, vt_dim);
    hipMemcpy(vt_out_c, vt_out_g, sizeof(bool), hipMemcpyDeviceToHost);
    EXPECT_TRUE(*vt_out_c);
    hipFree(vt_out1);
    hipFree(vt_out2);
    hipFree(vt_out_g);
    delete vt_out_c;
}

TEST_F(TestOps, TestOnesOp_gpu) {
    double* vt_out;
    double* vt_out_cpu = new double[vt_dim];
    hipMalloc(&vt_out, vt_dim * sizeof(double));
    ones_gpu_op()(device::gpu_device, vt_out, vt_dim);
    hipMemcpy(vt_out_cpu, vt_out, vt_dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < vt_dim; ++i) {
        EXPECT_EQ(vt_out_cpu[i], 1.0);
    }
    delete[] vt_out_cpu;
    hipFree(vt_out);
}

TEST_F(TestOps, TestEyeOp_gpu) {
    const int dim = 100;
    double* vt_out;
    double* vt_out_cpu = new double[dim * dim];
    hipMalloc(&vt_out, dim * dim * sizeof(double));
    eye_gpu_op()(device::gpu_device, vt_out, dim);
    hipMemcpy(vt_out_cpu, vt_out, dim * dim * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < dim; ++i) {
        for (int j = 0; j < dim; ++j) {
            if (i == j) {
                EXPECT_EQ(vt_out_cpu[i * dim + j], 1.0);
            } else {
                EXPECT_EQ(vt_out_cpu[i * dim + j], 0.0);
            }
        }
    }
}

TEST_F(TestOps, TestIm2ColOp_gpu) {
    /**
     * example img is 
     * [
     * [1, 2, 3;
     *  4, 5, 6;
     *  7, 8, 9],
     * [3, 2, 1;
     *  6, 5, 4;
     *  9, 8, 7]
     * ]
     * 
     * so data_im is
     * [1, 2, 3, 4, 5, 6, 7, 8, 9, 3, 2, 1, 6, 5, 4, 9, 8, 7]
     * 
     * im2col(img)(2X2 kernel, 1X1 stride, 0 padding) is
     * [
     *  1, 2, 4, 5, 3, 2, 6, 5;
     *  2, 3, 5, 6, 2, 1, 5, 4;
     *  4, 5, 7, 8, 6, 5, 9, 8;
     *  5, 6, 8, 9, 5, 4, 8, 7
     * ]
     * 
     * so data_col is
     * [1, 2, 4, 5, 3, 2, 6, 5, 2, 3, 5, 6, 2, 1, 5, 4, 4, 5, 7, 8, 6, 5, 9, 8, 5, 6, 8, 9, 5, 4, 8, 7]
     */
    int data_im[18] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 3, 2, 1, 6, 5, 4, 9, 8, 7};
    int data_col[32] = {0};
    int gt_col[32] = {1, 2, 4, 5, 3, 2, 6, 5, 2, 3, 5, 6, 2, 1, 5, 4, 4, 5, 7, 8, 6, 5, 9, 8, 5, 6, 8, 9, 5, 4, 8, 7};

    int* gdata_im;
    int* gdata_col;
    hipMalloc(&gdata_im, 18 * sizeof(int));
    hipMalloc(&gdata_col, 32 * sizeof(int));

    hipMemcpy(gdata_im, data_im, 18 * sizeof(int), hipMemcpyHostToDevice);
    im2col_gpu_op()(device::gpu_device, gdata_im, gdata_col, 2, 3, 3, 2, 2, 0, 0, 1, 1);
    hipMemcpy(data_col, gdata_col, 32 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 32; ++i) {
        EXPECT_EQ(data_col[i], gt_col[i]);
    }
}

TEST_F(TestOps, TestMaxPoolOp_gpu) {
    double data_im[32] = {
        // 1 channels
        -0.5752, 1.1023, 0.8327, -0.3337, 
        -0.0532, 0.8745, 1.4135, -0.4422, 
        -0.4538, 0.2952, 0.4086, -0.3135, 
        0.6764, 0.3422, -0.1896, 0.3065,
        // 2 channels 
        -0.3942, 1.3151, 0.5020, 0.7686, 
        -1.7310, 0.8545, -1.3705, -0.3178, 
        -2.5553, 1.1632, 0.4868, -0.1809,  
        0.0281, 1.2346, 0.3800, 0.2100
    };
    double gt_out[8] = {
        1.1023, 1.4135, 
        0.6764, 0.4086, 
        1.3151, 0.7686, 
        1.2346, 0.4868
    };
    int gt_mask[8] = {1, 2, 2, 0, 1, 1, 3, 0};

    double* d_data_im;
    double* d_data_out;
    int* d_mask_out;

    double data_out[8] = {0.0};
    int mask_out[8] = {0};
    
    hipMalloc(&d_data_im, 32 * sizeof(double));
    hipMalloc(&d_data_out, 8 * sizeof(double));
    hipMalloc(&d_mask_out, 8 * sizeof(int));

    hipMemcpy(d_data_im, data_im, 32 * sizeof(double), hipMemcpyHostToDevice);
    max_pool_gpu_op()(
        device::gpu_device, 
        d_data_out, 
        d_mask_out, 
        d_data_im, 
        1, 2, 
        4, 4, 
        2, 2, 
        0, 0, 
        2, 2
    );
    hipMemcpy(data_out, d_data_out, 8 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(mask_out, d_mask_out, 8 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 8; ++i) {
        EXPECT_NEAR(data_out[i], gt_out[i], 1e-4);
        EXPECT_EQ(mask_out[i], gt_mask[i]);
    }

    hipFree(d_data_im);
    hipFree(d_data_out);
    hipFree(d_mask_out);
}

TEST_F(TestOps, TestMaxPoolBackwardOp_gpu) {
    int mask_out[8] = {1, 2, 2, 0, 1, 1, 3, 0};
    double grad_out[8] = {1, 1, 1, 1, 1, 1, 1, 1};
    double grad_im[32] = {0};
    
    double gt_grad_im[32] = {
        0, 1, 0, 0, 
        0, 0, 1, 0, 
        0, 0, 1, 0, 
        1, 0, 0, 0, 
        0, 1, 0, 1, 
        0, 0, 0, 0, 
        0, 0, 1, 0, 
        0, 1, 0, 0
    };

    int* d_mask_out;
    double* d_grad_out;
    double* d_grad_im;
    
    hipMalloc(&d_mask_out, 8 * sizeof(int));
    hipMalloc(&d_grad_out, 8 * sizeof(double));
    hipMalloc(&d_grad_im, 32 * sizeof(double));

    hipMemcpy(d_mask_out, mask_out, 8 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_grad_out, grad_out, 8 * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_grad_im, 0, 32 * sizeof(double));  // 初始化为0

    max_pool_backward_gpu_op()(
        device::gpu_device, 
        d_grad_im,  
        d_mask_out,   
        d_grad_out,  
        1, 2,           
        4, 4,         
        2, 2,         
        0, 0,         
        2, 2          
    );

    hipMemcpy(grad_im, d_grad_im, 32 * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < 32; ++i) {
        EXPECT_NEAR(grad_im[i], gt_grad_im[i], 1e-4);
    }

    hipFree(d_mask_out);
    hipFree(d_grad_out);
    hipFree(d_grad_im);
}

int main(int argc, char** argv) {
std::cout << "run test for CORE::KERNELS::OPS::GPU" << std::endl << std::endl;
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
